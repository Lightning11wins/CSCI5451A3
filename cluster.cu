#include "hip/hip_runtime.h"
#include "cluster.h"

// Expects params to be initialized.
static inline double* read_points(char* filename, uint* num_points, uint* num_dims_ptr) {
    FILE* file = fopen(filename, "r");
    if (file == NULL) {
        perror("cluster.c: Fail - fopen()");
        exit(1);
    }

    check(fscanf(file, "%d %d", num_points, num_dims_ptr), "fscanf()");

    uint num_dims = *num_dims_ptr;
    double* points = (double*) malloc(*num_points * num_dims * sizeof(double));
    for (uint point = 0; point < *num_points; point++) {
        for (uint dim = 0; dim < num_dims; dim++) {
            check(fscanf(file, "%lf", &(points[index(point, dim)])), "fscanf()");
        }
    }

    check(fclose(file), "fclose()");

    return points;
}

static inline void write_clusters(uint* point_medoid_ids, uint num_points) {
    FILE *file = fopen(CLUSTER_OUTPUT_PATH, "w");
    if (file == NULL) {
        perror("cluster.c: Fail - fopen()");
        exit(1);
    }

    for (uint point_id = 0; point_id < num_points; point_id++) {
        fprintf(file, "%d\n", point_medoid_ids[point_id]);
    }

    check(fclose(file), "fclose()");
}

static inline void write_medoids(double* points, uint* medoids, uint num_medoids, uint num_dims) {
    FILE *file = fopen(MEDOID_OUTPUT_PATH, "w");
    if (file == NULL) {
        perror("cluster.c: Fail - fopen()");
        exit(1);
    }

    fprintf(file, "%d %d", num_medoids, num_dims);
    for (uint id = 0; id < num_medoids; id++) {
        fprintf(file, "\n%.4f", points[index(medoids[id], 0)]);
        for (uint dim = 1; dim < num_dims; dim++) {
            fprintf(file, " %.4f", points[index(medoids[id], dim)]);
        }
    }

    check(fclose(file), "fclose()");
}

static inline void print_points(double* points, uint num_points, uint num_dims) {
    printf("%dx%d\n", num_points, num_dims);
    for (uint i = 0; i < num_points; i++) {
        for (uint j = 0; j < num_dims; j++) {
            printf("%.1f ", points[index(i, j)]);
        }
        printf("\n");
    }
}

__device__ __host__
static inline void get_chunk(uint p, uint i, uint n, uint* start, uint* end) {
    // Slower version with branching, easier to understand.
    uint chunk_size = n / p, extra = n % p;
    *start = i * chunk_size + ((i < extra) ? i : extra);
    *end = *start + chunk_size + ((i < extra) ? 1 : 0);
    
    // Faster version, less readable.
    // int chunk_size = n / p, extra = n % p;
    // int offset = i * (chunk_size + 1) - (i >= extra) * (i - extra);
    // *start = offset;
    // *end = offset + chunk_size + (i < extra);
}

__global__
void assign_points_to_clusters(
    uint* point_medoid_ids,
    double* points,
    uint* num_points_ptr,
    uint* medoids,
    uint* num_medoids_ptr,
    uint* num_dims_ptr
) {
    // Calculate my point allocation.
    uint p = gridDim.x * blockDim.x;
    uint i = threadIdx.x + blockIdx.x * blockDim.x;
    uint n = *num_points_ptr;
    uint start, end;
    get_chunk(p, i, n, &start, &end);

    // Itterate through my points.
    for (uint point_id = start; point_id < end; point_id++) {
        double min_distance = INFINITY;
        uint closest_medoid_id = 1234567890;

        // Itterate through medoids to find the closest one.
        for (uint id = 0; id < *num_medoids_ptr; id++) {
            const double* point = points + point_id;
            const double* medoid = points + medoids[id];
            double distance = get_distance(point, medoid, *num_dims_ptr);
            if (distance < min_distance) {
                min_distance = distance;
                closest_medoid_id = id;
            }
        }

        // Update this point to be part of the closest cluster.
        point_medoid_ids[point_id] = closest_medoid_id;
    }
}

__global__
void get_cluster_sizes(
    double* point_cluster_sizes,
    double* points,
    uint* point_medoid_ids,
    uint* num_points_ptr,
    uint* num_dims_ptr
) {
    uint p = gridDim.x * blockDim.x;
    uint i = threadIdx.x + blockIdx.x * blockDim.x;
    uint n = *num_points_ptr;
    uint start, end;
    get_chunk(p, i, n, &start, &end);
    
    uint num_points = *num_points_ptr;
    uint num_dims = *num_dims_ptr;

    // Calculate the cluster size for each point if that point were the mediod
    for (uint point_id = 0; point_id < num_points; point_id++) {
        double total_distance = 0.0;
        uint point_count = 0;
        for (uint other_point_id = 0; other_point_id < num_points; other_point_id++) {
            if (point_medoid_ids[other_point_id] == point_medoid_ids[point_id]) {
                total_distance += get_distance(points + other_point_id, points + point_id, num_dims);
                point_count++;
            }
        }
        point_cluster_sizes[point_id] = total_distance / point_count;
    }
}

int main(int argc, char* argv[]) {
    if (argc <= 4) {
        char exe_name[PATH_MAX] = {0};
        get_exe_name(exe_name, PATH_MAX);

        fprintf(stderr, "Usage: ./%s <input_file> <num_medoids> <num_blocks> <num_threads_per_block>\n", exe_name);
        exit(1);
    }

    // Read the file.
    uint pts, dms;
    double* points = read_points(argv[1], &pts, &dms);
    const uint num_points = pts,
        num_dims = dms,
        num_medoids = parse_int(argv[2]),
        num_blocks = parse_int(argv[3]),
        num_threads_per_block = parse_int(argv[4]);

    // Error checking.
    if (num_points < num_medoids) {
        fprintf(stderr, "FAIL: Cannot cluster %d points with only %d medoids\n", num_points, num_medoids);
        exit(1);
    }

    printf("Clustering:\n  num_points: %d\n  num_dims: %d\n  num_medoids: %d\n  num_blocks: %d\n  num_threads_per_block: %d\n\n",
        num_points, num_dims, num_medoids, num_blocks, num_threads_per_block);

    // Start the clock.
    start_timer();

    // Select initial medoids.
    uint medoids[num_medoids];
    for (uint medoid_id = 0; medoid_id < num_medoids; medoid_id++) {
        medoids[medoid_id] = medoid_id; // Assign an id to each medoid.
    }

    // Define initial data.
    double average_cluster_size = INFINITY;
    uint point_medoid_ids[num_points];

    for (uint iteration = 0; iteration < 20; iteration++) {
        {   // Assign points to medoids.
            uint* gpu_point_medoid_ids;
            double* gpu_points;
            uint* gpu_num_points_ptr;
            uint* gpu_medoids;
            uint* gpu_num_clusters_ptr;
            uint* gpu_num_dims_ptr;

            check_cuda(hipMalloc((void**) &gpu_point_medoid_ids, num_points * sizeof(uint)),               "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_points,           num_points * num_dims * sizeof(double)), "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_num_points_ptr,   sizeof(uint)),                            "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_medoids,          num_medoids * sizeof(uint)),              "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_num_clusters_ptr, sizeof(uint)),                            "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_num_dims_ptr,     sizeof(uint)),                            "hipMalloc");

            check_cuda(hipMemcpy(gpu_point_medoid_ids, point_medoid_ids,  num_points * sizeof(uint),               TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_points,           points,            num_points * num_dims * sizeof(double), TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_num_points_ptr,   &num_points,       sizeof(uint),                            TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_medoids,          medoids,           num_medoids * sizeof(uint),              TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_num_clusters_ptr, &num_medoids,      sizeof(uint),                            TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_num_dims_ptr,     &num_dims,         sizeof(uint),                            TO_GPU), "cudaMemcpyTo");

            assign_points_to_clusters<<<num_blocks, num_threads_per_block>>>(
                gpu_point_medoid_ids,
                gpu_points,
                gpu_num_points_ptr,
                gpu_medoids,
                gpu_num_clusters_ptr,
                gpu_num_dims_ptr
            );
            
            check_cuda(hipMemcpy(point_medoid_ids, gpu_point_medoid_ids, num_points * sizeof(uint), FROM_GPU), "cudaMemcpyFrom");
            
            check_cuda(hipFree(gpu_point_medoid_ids), "hipFree");
            check_cuda(hipFree(gpu_points),           "hipFree");
            check_cuda(hipFree(gpu_num_points_ptr),   "hipFree");
            check_cuda(hipFree(gpu_medoids),          "hipFree");
            check_cuda(hipFree(gpu_num_clusters_ptr), "hipFree");
            check_cuda(hipFree(gpu_num_dims_ptr),     "hipFree");
        }

         // point_cluster_sizes: The size of this cluster if this the point with this id were the medoid.
        double point_cluster_sizes[num_points];
        for (uint i = 0; i < num_points; i++) point_cluster_sizes[i] = INFINITY; // Remove later
        
        {
            double* gpu_point_cluster_sizes;
            double* gpu_points;
            uint* gpu_point_medoid_ids;
            uint* gpu_num_points_ptr;
            uint* gpu_num_dims_ptr;
            
            check_cuda(hipMalloc((void**) &gpu_point_cluster_sizes, num_points * sizeof(double)),            "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_points,              num_points * num_dims * sizeof(double)), "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_point_medoid_ids,    num_points * sizeof(uint)),               "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_num_points_ptr,      sizeof(uint)),                            "hipMalloc");
            check_cuda(hipMalloc((void**) &gpu_num_dims_ptr,        sizeof(uint)),                            "hipMalloc");

            check_cuda(hipMemcpy(gpu_point_cluster_sizes, point_cluster_sizes, num_points * sizeof(double),            TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_points,              points,              num_points * num_dims * sizeof(double), TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_point_medoid_ids,    point_medoid_ids,    num_points * sizeof(uint),               TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_num_points_ptr,      &num_points,         sizeof(uint),                            TO_GPU), "cudaMemcpyTo");
            check_cuda(hipMemcpy(gpu_num_dims_ptr,        &num_dims,           sizeof(uint),                            TO_GPU), "cudaMemcpyTo");
            
            get_cluster_sizes<<<num_blocks, num_threads_per_block>>>(
                gpu_point_cluster_sizes,
                gpu_points,
                gpu_point_medoid_ids,
                gpu_num_points_ptr,
                gpu_num_dims_ptr
            );
            
            check_cuda(hipMemcpy(point_cluster_sizes, gpu_point_cluster_sizes, num_medoids * sizeof(double), FROM_GPU), "cudaMemcpyFrom");
            
            check_cuda(hipFree(gpu_point_cluster_sizes), "hipFree");
            check_cuda(hipFree(gpu_points),              "hipFree");
            check_cuda(hipFree(gpu_point_medoid_ids),    "hipFree");
            check_cuda(hipFree(gpu_num_points_ptr),      "hipFree");
            check_cuda(hipFree(gpu_num_dims_ptr),        "hipFree");
        }

        double medoid_sizes[num_points];
        for (uint point_id = 0; point_id < num_points; point_id++) {
            const uint medoid_id = point_medoid_ids[point_id];
            double size = point_cluster_sizes[point_id];
            if (size < medoid_sizes[medoid_id]) {
                medoid_sizes[medoid_id] = size;
                medoids[medoid_id] = point_id;
            }
        }

        double total_size = 0.0;
        for (uint i = 0; i < num_medoids; i++) total_size += medoid_sizes[i];
        double new_average_cluster_size = total_size / num_medoids;
        double dif = average_cluster_size - new_average_cluster_size;
        printf("Cluster size was %f, but is now %f (dif: %f). %d iterations remaining\n", average_cluster_size, new_average_cluster_size, dif, 20 - iteration);

        if (dif < 0) {
            // This should not be possible.
            printf("EXISTANCE BRINGS PAIN, PLEASE FREE ME!!\n");
            fflush(stdout);
        }

        if (dif < convergence_threshold) break;
        average_cluster_size = new_average_cluster_size;
    }

    // End the clock and print time.
    stop_timer();
    print_timer();

    // Write output data.
    write_clusters(point_medoid_ids, num_points);
    write_medoids(points, medoids, num_medoids, num_dims);

    // Cleanup main memory.
    free(points);

    return 0; // Success
}
